#include<stdio.h>
#include <hip/hip_runtime.h>


int main(){
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    if (deviceProp.globalL1CacheSupported) {
        printf("Global L1 cache supported\n");
    }
    else {
		printf("Global L1 cache NOT supported\n");
    }

    hipSharedMemConfig SharedMemConfig;
    hipDeviceGetSharedMemConfig(&SharedMemConfig);
    printf("SharedMemConfig = %d\n", SharedMemConfig);
    if(SharedMemConfig == hipSharedMemBankSizeEightByte){
		printf("cudaSharedMemBankSizeEightByte\n");
	}
	else if(SharedMemConfig == hipSharedMemBankSizeFourByte){
		printf("cudaSharedMemBankSizeFourByte\n");
	}
	else if(SharedMemConfig == hipSharedMemBankSizeDefault){
		printf("cudaSharedMemBankSizeDefault\n");
	}
	else{
		printf("cudaSharedMemBankSizeInvalid\n");
	}
}