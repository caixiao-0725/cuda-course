#include<stdio.h>
#include <hip/hip_runtime.h>


int main(){
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    if (deviceProp.globalL1CacheSupported) {
        printf("Global L1 cache supported\n");
    }
    else {
		printf("Global L1 cache NOT supported\n");
    }

    hipSharedMemConfig SharedMemConfig;
   
}