#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * An example of using a statically declared global variable (devData) to store
 * a floating-point value on the device.
 */

__device__ float devData;

__global__ void checkGlobalVariable()
{
    // display the original value
    printf("Device: the value of the global variable is %f\n", devData);

    unsigned int tid = threadIdx.x;
    volatile int temp[15];
    int a = 1;
    int b = 1;
    // alter the value
    devData += (a+b);
}

int main(void)
{
    // initialize the global variable
    float value = 3.14f;
    hipMemcpyToSymbol(HIP_SYMBOL(devData), &value, sizeof(float));
    printf("Host:   copied %f to the global variable\n", value);

    // invoke the kernel
    checkGlobalVariable<<<1, 1>>>();

    // copy the global variable back to the host
    hipMemcpyFromSymbol(&value, HIP_SYMBOL(devData), sizeof(float));
    printf("Host:   the value changed by the kernel to %f\n", value);

    hipDeviceReset();
    return EXIT_SUCCESS;
}