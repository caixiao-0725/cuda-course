#include "hip/hip_runtime.h"
#include<stdio.h>

#define imin(a,b) (a<b?a:b)

const int N = 100 * 1024 * 1024;
const int threadsPerBlock = 256;
const int blocksPerGrid = imin(32, (N + threadsPerBlock - 1) / threadsPerBlock);

__global__ void dot(int size, float *a, float *b, float *c) {
    __shared__ float cache[threadsPerBlock];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    float temp = 0;
    while (tid < size) {
        temp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }

    // set the cache values
    cache[cacheIndex] = temp;

    // synchronize threads in this block
    __syncthreads();

    // for reductions, threadsPerBlock must be a power of 2
    // because of the following code
    int i = blockDim.x / 2;
    while (i != 0) {
        if (cacheIndex < i)
            cache[cacheIndex] += cache[cacheIndex + i];
        __syncthreads();
        i /= 2;
    }

    if (cacheIndex == 0)
        c[blockIdx.x] = cache[0];
}

float malloc_test(int size) {
    hipEvent_t start, stop;
    float *a, *b, c, *partial_c;
    float *dev_a, *dev_b, *dev_partial_c;
    float elapsedTime;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    // allocate memory on the CPU side
    a = (float*) malloc(size * sizeof(float));
    b = (float*) malloc(size * sizeof(float));
    partial_c = (float*) malloc(blocksPerGrid * sizeof(float));

    // allocate the memory on the GPU
    hipMalloc((void**) &dev_a, size * sizeof(float));
    hipMalloc((void**) &dev_b, size * sizeof(float));
    hipMalloc((void**) &dev_partial_c, blocksPerGrid * sizeof(float));

    // fill in the host memory with data
    for (int i = 0; i < size; i++) {
        a[i] = i;
        b[i] = i * 2;
    }

    hipEventRecord(start, 0);
    // copy the arrays 'a' and 'b' to the GPU
    hipMemcpy(dev_a, a, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(float), hipMemcpyHostToDevice);

    dot<<<blocksPerGrid, threadsPerBlock>>>(size, dev_a, dev_b, dev_partial_c);
    // copy the array 'c' back from the GPU to the CPU
    hipMemcpy(partial_c, dev_partial_c, blocksPerGrid * sizeof(float),
            hipMemcpyDeviceToHost);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    // finish up on the CPU side
    c = 0;
    for (int i = 0; i < blocksPerGrid; i++) {
        c += partial_c[i];
    }

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_partial_c);

    // free memory on the CPU side
    free(a);
    free(b);
    free(partial_c);

    // free events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("result:  %f\n", c);

    return elapsedTime;
}

float cuda_pinned_alloc_test(int size) {
    hipEvent_t start, stop;
    float *a, *b, c, *partial_c;
    float *dev_a, *dev_b, *dev_partial_c;
    float elapsedTime;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    // allocate the memory on the CPU
    hipHostAlloc((void**) &a, size * sizeof(float),
            hipHostMallocWriteCombined | hipHostMallocMapped);
    hipHostAlloc((void**) &b, size * sizeof(float),
            hipHostMallocWriteCombined | hipHostMallocMapped);
    hipHostAlloc((void**) &partial_c, blocksPerGrid * sizeof(float),
            hipHostMallocMapped);

    // find out the GPU pointers
    hipHostGetDevicePointer(&dev_a, a, 0);
    hipHostGetDevicePointer(&dev_b, b, 0);
    hipHostGetDevicePointer(&dev_partial_c, partial_c, 0);

    // fill in the host memory with data
    for (int i = 0; i < size; i++) {
        a[i] = i;
        b[i] = i * 2;
    }

    hipEventRecord(start, 0);

    dot<<<blocksPerGrid, threadsPerBlock>>>(size, dev_a, dev_b, dev_partial_c);

    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    // finish up on the CPU side
    c = 0;
    for (int i = 0; i < blocksPerGrid; i++) {
        c += partial_c[i];
    }

    hipHostFree(a);
    hipHostFree(b);
    hipHostFree(partial_c);

    // free events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("result:  %f\n", c);

    return elapsedTime;
}

float cuda_host_alloc_test(int size) {
    hipEvent_t start, stop;
    float *aa, *bb, c, *partial_cc;
    float *dev_aa, *dev_bb, *dev_partial_cc;
    float elapsedTime;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    // allocate memory on the CPU side
    hipHostAlloc((void**) &aa, size * sizeof(*aa), hipHostMallocDefault);
    hipHostAlloc((void**) &bb, size * sizeof(*bb), hipHostMallocDefault);
    hipHostAlloc((void**) &partial_cc, size * sizeof(*partial_cc), hipHostMallocDefault);

    // allocate the memory on the GPU
    hipMalloc((void**) &dev_aa, size * sizeof(float));
    hipMalloc((void**) &dev_bb, size * sizeof(float));
    hipMalloc((void**) &dev_partial_cc, blocksPerGrid * sizeof(float));

    // fill in the host memory with data
    for (int i = 0; i < size; i++) {
        aa[i] = i;
        bb[i] = i * 2;
    }

    hipEventRecord(start, 0);
    // copy the arrays 'a' and 'b' to the GPU

    hipMemcpy(dev_aa, aa, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_bb, bb, size * sizeof(float), hipMemcpyHostToDevice);


    dot<<<blocksPerGrid, threadsPerBlock>>>(size, dev_aa, dev_bb, dev_partial_cc);
    // copy the array 'c' back from the GPU to the CPU
    hipMemcpy(partial_cc, dev_partial_cc, blocksPerGrid * sizeof(float),hipMemcpyDeviceToHost);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    // finish up on the CPU side
    c = 0;
    for (int i = 0; i < blocksPerGrid; i++) {
        c += partial_cc[i];
    }

    hipFree(dev_aa);
    hipFree(dev_bb);
    hipFree(dev_partial_cc);

    // free memory on the CPU side
    hipHostFree(aa);
    hipHostFree(bb);
    hipHostFree(partial_cc);

    // free events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("result:  %f\n", c);

    return elapsedTime;
}

int main(void) {
    hipDeviceProp_t prop;
    int whichDevice;
    hipGetDevice(&whichDevice);
    hipGetDeviceProperties(&prop, whichDevice);
    if (prop.canMapHostMemory != 1) {
        printf("Device can not map memory.\n");
        return 0;
    }
    float elapsedTime;

    hipSetDeviceFlags (hipDeviceMapHost);

    // try it with malloc
    elapsedTime = malloc_test(N);
    printf("cpu  time :  %3.1f ms\n", elapsedTime);

    // now try it with hipHostAlloc
    elapsedTime = cuda_pinned_alloc_test(N);
    printf("zero time :  %3.1f ms\n", elapsedTime);

    // now try it with hipHostAlloc
    elapsedTime = cuda_host_alloc_test(N);
    printf("pinned time:  %3.1f ms\n", elapsedTime);
}